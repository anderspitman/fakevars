#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>
#include <inttypes.h>

#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}


const uint64_t ALLELES_SIZE = 2;
const uint64_t BASE_SIZE = 2;
const uint64_t MIN_PHRED_QUAL = 0x21;
const uint64_t MAX_PHRED_QUAL = 0x7e;
const double ERROR_RATE = 0.02;
const uint64_t NUM_BASES = 4;
__device__ const char BASES[NUM_BASES] = { 'A', 'C', 'G', 'T' };

const uint64_t GENOTYPE_SIZE = 2;

__device__ uint64_t rand_in_range(uint64_t min, uint64_t max, hiprandState_t* state) {
        float rand = hiprand_uniform(state);
        return (rand * (max - min + 0.999999)) + min;
}

__device__ void gen_base_array_kernel(
        uint8_t* data,
        uint64_t depth,
        char* true_genotype,
        hiprandState* local_rand_state)
{
        //int tid = threadIdx.x + blockIdx.x * blockDim.x;
        //hiprandState local_rand_state = rand_state[tid];

        uint8_t* ptr;
        for (uint64_t i = 0; i < depth; i++) {
                ptr = (uint8_t*)&(data[i*BASE_SIZE]);
                float error = hiprand_uniform(local_rand_state);

                if (error < ERROR_RATE) {
                        // Simulate error by choosing random base
                        uint64_t idx = rand_in_range(0, NUM_BASES-1, local_rand_state);
                        ptr[0] = BASES[idx];
                }
                else {
                        // Sample randomly from true_genotype
                        uint64_t idx = rand_in_range(0, GENOTYPE_SIZE-1, local_rand_state);
                        //printf("%" PRIu64 "\n", idx);
                        ptr[0] = true_genotype[idx];
                }

                uint64_t qual = rand_in_range(MIN_PHRED_QUAL, MAX_PHRED_QUAL, local_rand_state);
                ptr[1] = qual;
        }
}

__device__ void gen_sample_kernel(
        uint8_t* data,
        uint64_t depth,
        char* locus_alleles,
        hiprandState* local_rand_state)
{

        uint64_t base_1_idx = rand_in_range(0, ALLELES_SIZE-1, local_rand_state);
        uint64_t base_2_idx = rand_in_range(0, ALLELES_SIZE-1, local_rand_state);

        // true_genotype
        data[0] = locus_alleles[base_1_idx];
        data[1] = locus_alleles[base_2_idx];

        gen_base_array_kernel(data + GENOTYPE_SIZE, depth, (char*)data, local_rand_state);
}

__device__ void gen_locus_kernel(
        uint8_t* data,
        uint64_t num_samples,
        uint64_t depth,
        hiprandState* local_rand_state)
{

        const uint64_t base_array_size = depth*BASE_SIZE;
        const uint64_t sample_size = GENOTYPE_SIZE + base_array_size;

        auto base_1 = rand_in_range(0, NUM_BASES-1, local_rand_state);
        auto base_2 = rand_in_range(0, NUM_BASES-1, local_rand_state);

        // alleles
        data[0] = BASES[base_1];
        data[1] = BASES[base_2];

        uint8_t* sample_ptr = &data[ALLELES_SIZE];

        for (uint64_t i = 0; i < num_samples; i++) {
                gen_sample_kernel(sample_ptr, depth, (char*)data, local_rand_state);
                sample_ptr += sample_size;
        }
}

__global__ void gen_loci_kernel(
        uint8_t* all_data,
        uint64_t num_loci,
        uint64_t num_samples,
        uint64_t depth,
        uint64_t rand_seed,
        hiprandState* rand_state)
{
        uint64_t tid = threadIdx.x + blockIdx.x * blockDim.x;


        if (tid > num_loci - 1) {
                //printf("ditching tid: %llu\n", tid);
                return;
        }

        hiprand_init(rand_seed, tid, 0, &rand_state[tid]);

        hiprandState local_rand_state = rand_state[tid];

        const uint64_t base_array_size = depth*BASE_SIZE;
        const uint64_t sample_size = GENOTYPE_SIZE + base_array_size;
        const uint64_t locus_size = ALLELES_SIZE + sample_size*num_samples;

        const uint64_t data_idx = tid*locus_size;
        uint8_t* data = &all_data[data_idx];

        for (uint64_t i = 0; i < locus_size; i++) {
                data[i] = 232;
        }

        auto base_1 = rand_in_range(0, NUM_BASES-1, &local_rand_state);
        auto base_2 = rand_in_range(0, NUM_BASES-1, &local_rand_state);

        // alleles
        data[0] = BASES[base_1];
        data[1] = BASES[base_2];

        uint8_t* sample_ptr = &data[ALLELES_SIZE];

        for (uint64_t i = 0; i < num_samples; i++) {
                gen_sample_kernel(sample_ptr, depth, (char*)data, &local_rand_state);
                sample_ptr += sample_size;
        }
}

void gen_data(uint64_t num_loci, uint64_t num_samples, uint64_t depth, uint8_t** d_data, uint64_t *size) {

        const uint64_t n_blocks = ceil(num_loci/256.0);
        const uint64_t n_threads = 256;
        const uint64_t seed = time(NULL);

        hiprandState* d_rand_states;

        hipMalloc(&d_rand_states, num_loci*sizeof(hiprandState));
        cudaCheckError();

        const uint64_t base_array_size = depth*BASE_SIZE;
        const uint64_t sample_size = GENOTYPE_SIZE + base_array_size;
        const uint64_t locus_size = ALLELES_SIZE + sample_size*num_samples;
        const uint64_t total_size = locus_size*num_loci;

        *size = total_size;

        hipMalloc(d_data, *size);
        cudaCheckError();

        gen_loci_kernel<<<n_blocks, n_threads>>>(*d_data, num_loci, num_samples, depth, seed, d_rand_states);
        hipDeviceSynchronize();
}


void gen_data_gpu(uint64_t num_loci, uint64_t num_samples, uint64_t depth, uint8_t** h_data, uint64_t *size) {

        uint8_t* d_data;

        gen_data(num_loci, num_samples, depth, &d_data, size);

        hipHostMalloc(h_data, *size);
        cudaCheckError();

        hipMemcpy(*h_data, d_data, *size, hipMemcpyDeviceToHost);
        cudaCheckError();
}
