
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>
#include <inttypes.h>

#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}


const uint64_t ALLELES_SIZE = 2;
const uint64_t BASE_SIZE = 2;
const uint64_t MIN_PHRED_QUAL = 0x21;
const uint64_t MAX_PHRED_QUAL = 0x7e;
const double ERROR_RATE = 0.02;
const uint64_t NUM_BASES = 4;
__device__ const char BASES[NUM_BASES] = { 'A', 'C', 'G', 'T' };

const uint64_t GENOTYPE_SIZE = 2;

__device__ uint64_t rand_in_range(uint64_t min, uint64_t max, hiprandState_t* state) {
        float rand = hiprand_uniform(state);
        return (rand * (max - min + 0.999999)) + min;
}

__global__ void init_rand_states(hiprandState* state, uint64_t seed) {
            int tid = threadIdx.x + blockIdx.x * blockDim.x;
            hiprand_init(seed, tid, 0, &state[tid]);
}

__device__ void gen_base_array_kernel(
        uint8_t* data,
        uint64_t depth,
        char* true_genotype,
        hiprandState* local_rand_state)
{
        //int tid = threadIdx.x + blockIdx.x * blockDim.x;
        //curandState local_rand_state = rand_state[tid];

        uint8_t* ptr;
        for (uint64_t i = 0; i < depth; i++) {
                ptr = (uint8_t*)&(data[i*BASE_SIZE]);
                float error = hiprand_uniform(local_rand_state);

                if (error < ERROR_RATE) {
                        // Simulate error by choosing random base
                        uint64_t idx = rand_in_range(0, NUM_BASES-1, local_rand_state);
                        ptr[0] = BASES[idx];
                }
                else {
                        // Sample randomly from true_genotype
                        uint64_t idx = rand_in_range(0, GENOTYPE_SIZE-1, local_rand_state);
                        //printf("%" PRIu64 "\n", idx);
                        ptr[0] = true_genotype[idx];
                }

                uint64_t qual = rand_in_range(MIN_PHRED_QUAL, MAX_PHRED_QUAL, local_rand_state);
                ptr[1] = qual;
        }
}

__device__ void gen_sample_kernel(
        uint8_t* data,
        uint64_t depth,
        char* locus_alleles,
        hiprandState* local_rand_state)
{

        uint64_t base_1_idx = rand_in_range(0, ALLELES_SIZE-1, local_rand_state);
        uint64_t base_2_idx = rand_in_range(0, ALLELES_SIZE-1, local_rand_state);

        // true_genotype
        data[0] = locus_alleles[base_1_idx];
        data[1] = locus_alleles[base_2_idx];

        gen_base_array_kernel(data + GENOTYPE_SIZE, depth, (char*)data, local_rand_state);
}

__device__ void gen_locus_kernel(
        uint8_t* data,
        uint64_t num_samples,
        uint64_t depth,
        hiprandState* local_rand_state)
{

        const uint64_t base_array_size = depth*BASE_SIZE;
        const uint64_t sample_size = GENOTYPE_SIZE + base_array_size;

        auto base_1 = rand_in_range(0, NUM_BASES-1, local_rand_state);
        auto base_2 = rand_in_range(0, NUM_BASES-1, local_rand_state);

        // alleles
        data[0] = BASES[base_1];
        data[1] = BASES[base_2];

        uint8_t* sample_ptr = &data[ALLELES_SIZE];

        for (uint64_t i = 0; i < num_samples; i++) {
                gen_sample_kernel(sample_ptr, depth, (char*)data, local_rand_state);
                sample_ptr += sample_size;
        }
}

__global__ void gen_loci_kernel(
        uint8_t* data,
        uint64_t num_loci,
        uint64_t num_samples,
        uint64_t depth,
        hiprandState* rand_state)
{
        int tid = threadIdx.x + blockIdx.x * blockDim.x;
        hiprandState local_rand_state = rand_state[tid];

        const uint64_t base_array_size = depth*BASE_SIZE;
        const uint64_t sample_size = GENOTYPE_SIZE + base_array_size;
        const uint64_t locus_size = ALLELES_SIZE + sample_size*num_samples;

        uint8_t* ptr = data;
        for (uint64_t i = 0; i < num_loci; i++) {
                gen_locus_kernel(ptr, num_samples, depth, &local_rand_state);
                ptr += locus_size;
        }
}


uint8_t* gen_data(uint64_t num_loci, uint64_t num_samples, uint64_t depth) {
        const uint64_t n_blocks = 256;
        const uint64_t n_threads = 256;
        const uint64_t seed = 9999;

        hiprandState* d_rand_states;
        hipError_t err;

        err = hipMalloc(&d_rand_states, n_blocks*n_threads*sizeof(hiprandState));
        assert(err == hipSuccess);

        init_rand_states<<<n_blocks, n_threads>>>(d_rand_states, seed);
        hipDeviceSynchronize();


        uint8_t* h_data;

        const uint64_t base_array_size = depth*BASE_SIZE;
        const uint64_t sample_size = GENOTYPE_SIZE + base_array_size;
        const uint64_t locus_size = ALLELES_SIZE + sample_size*num_samples;
        const uint64_t total_size = locus_size*num_loci;

        const uint64_t size = total_size;

        err = hipHostMalloc(&h_data, size, hipHostMallocDefault);
        assert(err == hipSuccess);

        uint8_t* d_data;
        err = hipMalloc(&d_data, size);
        assert(err == hipSuccess);

        char h_locus_alleles[2] = { 'A', 'T' };

        char* d_locus_alleles;
        hipMalloc(&d_locus_alleles, ALLELES_SIZE);
        cudaCheckError();
        hipMemcpy(d_locus_alleles, h_locus_alleles, GENOTYPE_SIZE, hipMemcpyHostToDevice);
        cudaCheckError();

        gen_loci_kernel<<<n_blocks, n_threads>>>(d_data, num_loci, num_samples, depth, d_rand_states);
        hipDeviceSynchronize();

        err = hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);
        cudaCheckError();

        return h_data;
}


